#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <sstream>
#include <utility>
#include <unordered_map>
#include <hip/hip_runtime.h>


#include <chrono>
#include <vector>
#include <assert.h>
#include <math.h>

#define NUM_STREAMS 16



// This is first version of the gpu implementation
// This version is just for testing sub-parts of the xnor convolution
constexpr std::pair<int, int> register_size(8, 4);
constexpr int nTPB = 256;

template <typename T>
struct matrix1d {
	int lenght;
	T *arr;
};

template <typename T>
struct matrix2d {
	int row;
	int col;
	T *arr;
};

template <typename T>
struct matrix3d {
	int row;
	int col;
	int channel;
	T **arr;
};

template <typename T>
struct weight4d{
	int row;
	int col;
	int channel_in;
	int channel_out;
	T **arr;
};


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

std::pair<int, int> find_binary_size(std::pair<int, int>input_size,  std::pair<int, int>kernel_size){
	int size_x = ceil((input_size.first - register_size.first)
						/static_cast<double>(register_size.first + 1 - kernel_size.first) + 1);
	int size_y = ceil((input_size.second - register_size.second )
						/static_cast<double>(register_size.second + 1 - kernel_size.second) + 1);
	if (size_x < 0)
		size_x = 1;
	if (size_y < 0)
		size_y = 1;
	return std::make_pair(size_x, size_y);
}

size_t choose_block_size(size_t val){
  if (val >= nTPB) return nTPB;
  if (val <= 32) return 32;
  val = (val >> 1) | val;
  val = (val >> 2) | val;
  val = (val >> 4) | val;
  val = (val >> 8) | val;
  val = (val >> 16) | val;
  val++;
  return val;
}

template<typename T>
void __global__ zeroPadding(matrix2d<T> *input_mat, matrix2d<T>* output_mat,  int kernel_row, int kernel_col)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int index_x = (idx % output_mat->col) - (kernel_col - 1)/ 2;
	int index_y = (idx/ output_mat->col) - (kernel_row - 1)/ 2;
	if(index_x > 0 || index_y>0 )
	{
		if( index_x< input_mat->col || index_y < input_mat->row)
		{
			output_mat[idx] = input_mat[index_y * output_mat->col + index_x];
		}
	}
	else output_mat[idx] = 0;
}
template<typename T>
void __global__ zeroPadding(T * input_mat, T * output_mat,  int kernel_row, int kernel_col, int input_row, int input_col, int output_row, int output_col)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int index_x = (idx % output_col) - (kernel_row - 1)/ 2;
	int index_y = (idx/output_col) - (kernel_col - 1)/ 2;
	if (idx < output_row * output_col)
	{
		if(index_x > 0 && index_y>0 )
		{
			if( index_x< input_col && index_y < input_row)
			{
				output_mat[idx] = input_mat[index_y * input_col + index_x];
			}
		}
		else output_mat[idx] = 0;
	}
}

template<typename T>
 __device__ void to_binary_register(
	const T &idata,
	unsigned int &odata,
	 int *output_location)
{
	int sign = (idata > 0) - (idata < 0);
	const unsigned int pozitive = 1;
	const unsigned int negative = 0;
	//int count = output_location[1] * register_size.second  + output_location[0];
	//assert(count < register_size.second * register_size.first);
	if (sign > -1)
	{
		odata = pozitive<<(output_location[1] * register_size.first  + output_location[0]) | odata;
	}
	else
	{
		odata = negative<<(output_location[1] * register_size.first  + output_location[0]) | odata;
	}
}

template<typename T>
void __global__  to_binary_matrix(
	const T *  d_idata,
	unsigned int *  d_odata,
	const int row, const int b_row,
	const int col, const int b_col,
	const int kernel_row = 3, const int kernel_col = 3)
{
	// Each thread will store a size = 32 array inside their single register
	int idx = threadIdx.x+blockDim.x*blockIdx.x; //register IDX
	// n*(regsiter_size - kernel_size)
	if (idx < (b_row * b_col))
	{
		int input_index[] = {(idx%b_col) * (register_size.first - kernel_col), (idx /b_col ) * (register_size.second - kernel_row)};
		int data_idx = input_index[0] + (input_index[1] * row);
		//int input_index[] = {data_idx%row, data_idx/col, data_idx/(row*col)}; // from start of array , (x, y, z)
		int register_location[] = {0, 0};
		unsigned int local_register;
		for (int j=0; register_size.second>j; j++)
		{
			for (int i=0; register_size.first>i; i++)
			{
				to_binary_register<T>(d_idata[data_idx], local_register, register_location);
				++data_idx;
				input_index[0] += 1;
				register_location[0] += 1;
				if (input_index[0] == col) break;
			}
			data_idx = data_idx + col - register_location[0];
			input_index[1] += 1;
			input_index[0] = (idx%b_col) * (register_size.first - kernel_col);
			register_location[0] = 0;
			register_location[1] += 1;
			if (input_index[1] == row) break;
		}
		d_odata[idx] = local_register;
	}
}
void __global__ binaryConv2d(
		const unsigned int * binary_mat,
		unsigned int * output_mat,
		const unsigned int *weight_matrix,
		int binary_row, int binary_col,
		int kernel_row, int kernel_col,
		int output_row, int output_col
		)
{

	int idx = threadIdx.x +blockDim.x*blockIdx.x; //binary Cell id
	int conv_per_row = register_size.second - (kernel_row - 1);
	int conv_per_column = register_size.first - (kernel_col - 1);
	int output_index_x = (idx % binary_col) * conv_per_column;
	int output_index_y = (idx / binary_col) * conv_per_row;
	//return;
	if (idx < binary_row * binary_col)
	{
	unsigned int register_buffer = binary_mat[idx];
	if ( (output_index_x + conv_per_column) > output_col)
	{
		conv_per_column = output_col - output_index_x;
	}
	if ( (output_index_y + conv_per_row) > output_row)
	{
		conv_per_row = output_row - output_index_y;
	}

	unsigned int mask = std::pow(2, kernel_col) - 1;

	for (int j=1; kernel_row > j; j++)
	{
		mask = (mask<<register_size.first) | static_cast<unsigned int>(std::pow(2, kernel_col) - 1);
	}

	unsigned int shifter = 0;
	for (int j=0; conv_per_row>j; ++j)
	{
		for (int i=0; conv_per_column>i; ++i)
		{

			output_mat[(output_index_y+j)*output_col + output_index_x + i] = (~(register_buffer>>shifter) ^ (weight_matrix[0]) ) & mask;
			++shifter;
		}
		// Check if register is not fully filled,
		// if not add shifter the missing shift amount
		shifter += register_size.second - conv_per_column;
	}
	}

}

void __global__ binary2int(unsigned int *input_mat, int matrix_row, int matrix_col, int kernel_row, int kernel_col)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < matrix_row* matrix_col)
	{
		unsigned int mask = 1;
		unsigned int shifter = 0;
		int buffer = 0;
		unsigned int data = input_mat[idx];
		for (int j=0; kernel_row>j; ++j)
		{
			for(int i=0; kernel_col>i; ++i)
			{
				buffer += (data >> shifter) & mask;
				++shifter;
			}
			shifter += register_size.first - kernel_col;
		}
		input_mat[idx] = 2 * buffer - (kernel_row * kernel_col);
	}
}

void __global__ kernel_reduce_sum(
		const unsigned int * __restrict__  d_idata,
		float * __restrict__ d_odata,
        const int col,
        const int row,
        const int channel)
{
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < (col * row)){
	  int tidx = idx;
	  float tsum = 0;
	  for (int i = 0; i < channel; i++)
	  {
		tsum += static_cast<float>(d_idata[tidx]);
		tidx += row * col;
	  }
	  d_odata[idx] = tsum / static_cast<float>(channel);
	}
}
// A single Xnor convolution,
// Inputs are input float matrix and weight tensor;  output as float output matrix
// There are two main part in Xnor convolution that can be done in concurrently
// Finding K matrix, and binary xnor convolution.
// Then convolution Result and K matrix can multiply in elementwise,
// Final result will be obtained by multiplying by alpha scalar.

matrix2d<unsigned int> floatMat2BinaryMat(matrix2d<float> &d_input_matrix, int kernel_col, int kernel_row, hipStream_t streamID = 0)
{
	hipStreamCreate ( &streamID) ;
	matrix2d<unsigned int> d_output_matrix;
	auto binary_size = find_binary_size(std::make_pair(d_input_matrix.col, d_input_matrix.row), std::make_pair(kernel_col, kernel_row));
	hipMalloc(&d_output_matrix.arr, binary_size.first * binary_size.second *sizeof(unsigned int));
	auto block_size = choose_block_size(binary_size.first * binary_size.second);
	d_output_matrix.col = binary_size.first;
	d_output_matrix.row = binary_size.second;
	to_binary_matrix<<<(d_output_matrix.row * d_output_matrix.col + block_size - 1)/ block_size , block_size, 0, streamID>>>
			(d_input_matrix.arr, d_output_matrix.arr, d_input_matrix.row, d_output_matrix.row, d_input_matrix.col, d_output_matrix.col);
	return d_output_matrix;
}




matrix3d<float> xnor_convolution_v1(matrix3d<float> &h_input_tensor, weight4d<float> &h_weight_tensor, bool padding=true)
{
		// Use cudaMallocHost
		//cudaStream_t streams[NUM_STREAMS];
		//for (int i = 0; i < NUM_STREAMS; ++i) { cudaStreamCreate(&streams[i]); }

	matrix3d<float> h_output_tensor;
	h_output_tensor.col = h_input_tensor.col;
	h_output_tensor.row = h_input_tensor.row;
	h_output_tensor.channel = h_weight_tensor.channel_out;
	h_output_tensor.arr = new float *[h_output_tensor.channel]();
	for(int i=0; h_output_tensor.channel > i; ++i)
	{
		h_output_tensor.arr[i] = new float [h_output_tensor.row * h_output_tensor.col];
	}


	for (int j=0; j < h_weight_tensor.channel_out; ++j )
	{
		unsigned int **h_channel_outputs = new unsigned int*[h_weight_tensor.channel_in]();
		for (int i=0; i<h_weight_tensor.channel_in; ++i)
		{
			h_channel_outputs[i] = new unsigned int[h_input_tensor.row * h_input_tensor.row];
		}
		for (int i=0; i< h_weight_tensor.channel_in; ++i)
		{
			hipEvent_t start, stop;
			hipEvent_t start1, stop1;
			hipEvent_t start2, stop2;
			hipEvent_t start3, stop3;
			hipEventCreate(&start2);
			hipEventCreate(&stop2);
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventCreate(&start1);
			hipEventCreate(&stop1);
			hipEventCreate(&start3);
			hipEventCreate(&stop3);
			float milliseconds = 0;
			matrix2d<float> d_input_matrix;
			d_input_matrix.col = h_input_tensor.col;
			d_input_matrix.row = h_input_tensor.row;
			hipMalloc((void **)&d_input_matrix.arr, sizeof(float) * d_input_matrix.col* d_input_matrix.row);
			hipMemcpy(d_input_matrix.arr, h_input_tensor.arr[i], d_input_matrix.col* d_input_matrix.row * sizeof(float), hipMemcpyHostToDevice);
			matrix2d<float> d_weight_matrix;
			d_weight_matrix.col = h_weight_tensor.col;
			d_weight_matrix.row = h_weight_tensor.row;
			hipMalloc((void **)&d_weight_matrix.arr, sizeof(float) * d_weight_matrix.col * d_weight_matrix.row);
			hipMemcpy(d_weight_matrix.arr, h_weight_tensor.arr[j*h_weight_tensor.channel_in + i], sizeof(float) * d_weight_matrix.col * d_weight_matrix.row, hipMemcpyHostToDevice);

			matrix2d<float> d_padded_matrix;
			d_padded_matrix.col = d_input_matrix.col + h_weight_tensor.col - 1;
			d_padded_matrix.row = d_input_matrix.row + h_weight_tensor.row - 1;
			hipMalloc((void **)&d_padded_matrix.arr, d_padded_matrix.col * d_padded_matrix.row * sizeof(float));
			auto block_size = choose_block_size(d_padded_matrix.row * d_padded_matrix.col);
			hipEventRecord(start3, 0);
			zeroPadding<float><<<(d_padded_matrix.row * d_padded_matrix.col + block_size - 1)/ block_size , block_size>>>(d_padded_matrix.arr, d_padded_matrix.arr, h_weight_tensor.row, h_weight_tensor.col, d_input_matrix.row, d_input_matrix.col, d_padded_matrix.row, d_padded_matrix.col);
			hipEventRecord(stop3, 0);
			hipEventSynchronize(stop3);
			hipEventElapsedTime(&milliseconds, start3, stop3);
			std::cout<<"ZeroPadding Time= "<< milliseconds<<std::endl;
			hipFree(d_input_matrix.arr);
			hipEventRecord(start, 0);
			auto d_binary_input_matrix = floatMat2BinaryMat(d_padded_matrix, h_weight_tensor.row, h_weight_tensor.col);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout<<"Integer to binary conversion Time= "<< milliseconds<<std::endl;
			hipFree(d_padded_matrix.arr);
			auto d_binary_weight_matrix = floatMat2BinaryMat(d_weight_matrix, h_weight_tensor.row, h_weight_tensor.col);
			hipFree(d_weight_matrix.arr);
			block_size = choose_block_size(d_binary_input_matrix.col * d_binary_input_matrix.row);
			matrix2d<unsigned int> d_binary_output_matrix;
			d_binary_output_matrix.col = h_input_tensor.col;
			d_binary_output_matrix.row = h_input_tensor.row;
			hipMalloc((void **)&d_binary_output_matrix.arr, d_binary_output_matrix.col * d_binary_output_matrix.row * sizeof(float));
			hipEventRecord(start1, 0);
			binaryConv2d<<<(d_binary_input_matrix.row * d_binary_input_matrix.col + block_size - 1)/ block_size ,block_size>>>(d_binary_input_matrix.arr, d_binary_output_matrix.arr, d_binary_weight_matrix.arr,
																												d_binary_input_matrix.row, d_binary_input_matrix.col,
																												d_weight_matrix.row, d_weight_matrix.col,
																												d_binary_output_matrix.row, d_binary_output_matrix.col);
			hipEventRecord(stop1, 0);
			hipEventSynchronize(stop1);
			hipEventElapsedTime(&milliseconds, start1, stop1);
			std::cout<<"Convolution Time= "<< milliseconds<<std::endl;
			block_size = choose_block_size(d_binary_output_matrix.col * d_binary_output_matrix.row);
			hipEventRecord(start2, 0);
			binary2int<<<(d_binary_output_matrix.row * d_binary_output_matrix.col + block_size - 1)/ block_size ,block_size>>>(d_binary_output_matrix.arr, d_binary_output_matrix.row, d_binary_output_matrix.col, d_weight_matrix.row, d_weight_matrix.col);
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&milliseconds, start2, stop2);
			std::cout<<"Binary to integer conversion Time= "<< milliseconds<<std::endl;
			hipMemcpy(h_channel_outputs[i], d_binary_output_matrix.arr, sizeof(unsigned int) * d_binary_output_matrix.row * d_binary_output_matrix.col, hipMemcpyDeviceToHost);
			hipFree(d_binary_output_matrix.arr);
			hipEventDestroy(start);
			hipEventDestroy(stop);
			hipEventDestroy(start1);
			hipEventDestroy(stop1);
			hipEventDestroy(start2);
			hipEventDestroy(stop2);
		}
		matrix2d<float> d_output_matrix;
		d_output_matrix.col = h_output_tensor.col;
		d_output_matrix.row = h_output_tensor.row;
		auto block_size = choose_block_size(d_output_matrix.col * d_output_matrix.row);
		unsigned int *buffer = new unsigned int[h_output_tensor.col * h_output_tensor.row * h_weight_tensor.channel_in];
		memcpy(buffer, h_channel_outputs, sizeof(h_channel_outputs));
		unsigned int *d_channel_outputs;
		hipMalloc((void**)&d_channel_outputs, sizeof(unsigned int) * h_output_tensor.row * h_output_tensor.col * h_weight_tensor.channel_in);
		hipMalloc((void **)&d_output_matrix.arr, sizeof(float) * h_output_tensor.row * h_output_tensor.col);
		hipMemcpy(d_channel_outputs, buffer, h_output_tensor.col * h_output_tensor.row * h_weight_tensor.channel_in * sizeof(unsigned int), hipMemcpyHostToDevice);
		kernel_reduce_sum<<<(d_output_matrix.row * d_output_matrix.col + block_size - 1)/ block_size, block_size>>>(d_channel_outputs, d_output_matrix.arr,
																													d_output_matrix.col, d_output_matrix.row, h_output_tensor.channel);
		hipMemcpy(h_output_tensor.arr[j], d_output_matrix.arr, sizeof(float) * h_output_tensor.row * h_output_tensor.col, hipMemcpyDeviceToHost );
		hipFree(d_channel_outputs);
		hipFree(d_output_matrix.arr);

	}



	return h_output_tensor;
}

int main()
{
	int row = 512;
	int col = 512;
	int kernel_row = 3;
	int kernel_col = 3;

	int channel_in = 64;
	int channel_out = 1;
	matrix3d<float> input_tensor;
	weight4d<float> weight_tensor;
	input_tensor.row = row;
	input_tensor.col = col;
	input_tensor.channel = channel_in;
	// Init Matrices
	input_tensor.arr = new float *[input_tensor.channel]();
	for(int i=0; input_tensor.channel > i; ++i)
	{
		input_tensor.arr[i] = new float [input_tensor.row * input_tensor.col];
	}
	weight_tensor.row = kernel_row;
	weight_tensor.col = kernel_col;
	weight_tensor.channel_in = channel_in;
	weight_tensor.channel_out = channel_out;
	weight_tensor.arr = new float *[weight_tensor.channel_in * weight_tensor.channel_out]();
	for(int i=0; (weight_tensor.channel_in * weight_tensor.channel_out) > i ; ++i)
	{
		weight_tensor.arr[i] = new float [weight_tensor.row * weight_tensor.col];
	}
	bool padding = true;
	// Default Values
	for(int i=0; input_tensor.channel > i; ++i)
	{
		for (int j=0; input_tensor.col * input_tensor.row> j; ++j)
		{
			input_tensor.arr[i][j] = (rand() % 50) -0;
		}
	}
	for(int i=0; weight_tensor.channel_in * weight_tensor.channel_out > i; ++i)
	{
		for (int j=0; weight_tensor.col * weight_tensor.row> j; ++j)
		{
			weight_tensor.arr[i][j] = (rand() % 50) -0;
		}
	}
	// A sample layer

	matrix3d<float> output_matrix = xnor_convolution_v1(input_tensor, weight_tensor, padding);

	for (int i = 0; i < input_tensor.channel; i++)
	{
	    delete[] input_tensor.arr[i];
	}
	delete[] input_tensor.arr;

	for (int i = 0; i < output_matrix.channel; i++)
	{
	    delete[] output_matrix.arr[i];
	}
	delete[] output_matrix.arr;

	for (int i = 0; i < weight_tensor.channel_in * weight_tensor.channel_out; i++)
	{
	    delete[] weight_tensor.arr[i];
	}
	delete[] weight_tensor.arr;

	return 0;
}



