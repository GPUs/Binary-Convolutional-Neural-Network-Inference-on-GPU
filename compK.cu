#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "stdlib.h"
#include "stdio.h"
#include "iostream"
#include "math.h"


#define BLOCKSIZE 32
#define TILE_WIDTH 16
#define maskCols 3
#define maskRows 3
#define w (TILE_WIDTH + maskCols -1)


//mask in constant memory
__constant__ float deviceMaskData[maskRows * maskCols];


__global__ void prepareInputKernel(float* InputImageData, const float* __restrict__ kernel,
    float* outputImageData, int channels, int width, int height) {

    __shared__ float N_ds[w][w];	//block of share memory


    // allocation in shared memory of image blocks
    int maskRadius = maskRows / 2;
    for (int k = 0; k < channels; k++) {
        int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
        int destY = dest / w;     //col of shared memory
        int destX = dest % w;		//row of shared memory
        int srcY = blockIdx.y * TILE_WIDTH + destY - maskRadius;  //row index to fetch data from input image
        int srcX = blockIdx.x * TILE_WIDTH + destX - maskRadius;	//col index to fetch data from input image
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = InputImageData[(srcY * width + srcX) * channels + k];
        else
            N_ds[destY][destX] = 0;


        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w;
        destX = dest % w;
        srcY = blockIdx.y * TILE_WIDTH + destY - maskRadius;
        srcX = blockIdx.x * TILE_WIDTH + destX - maskRadius;
        if (destY < w) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = InputImageData[(srcY * width + srcX) * channels + k];
            else
                N_ds[destY][destX] = 0;
        }

        __syncthreads();


        //compute kernel convolution
        float accum = 0;
        int y, x;
        for (y = 0; y < maskCols; y++)
            for (x = 0; x < maskRows; x++)
                accum += N_ds[threadIdx.y + y][threadIdx.x + x] * deviceMaskData[y * maskCols + x];

        y = blockIdx.y * TILE_WIDTH + threadIdx.y;
        x = blockIdx.x * TILE_WIDTH + threadIdx.x;
        if (y < height && x < width)
            outputImageData[(y * width + x) * channels + k] = accum;
        __syncthreads();

    }
}


__global__ void addMatrices(float* matrices, float* result, int channels, int width, int height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < height * width) {
        float sum = 0;
        for (int i = 0; i < channels; i++) {
            sum += matrices[i * width * height + idx];
        }
        result[idx] = sum;
    }
}


void clearMemory(float* p, int size) {
    hipMemset(p, 0, size);
    hipFree(p);
}


void print(float* deviceOutputImageData, int imageHeight, int imageWidth) {
    float* hostOutputImageData;
    hostOutputImageData = new float[imageHeight * imageWidth];
    hipMemcpy(hostOutputImageData, deviceOutputImageData,
    imageWidth * imageHeight * sizeof(float),
    hipMemcpyDeviceToHost);

    for (int i = 0; i < imageHeight * imageWidth; i++) {
        printf("%f, ", hostOutputImageData[i]);
    }

    free(hostOutputImageData);
}

float* prepareInput(float *deviceInputImageData, int imageHeight, int imageWidth, int imageChannels) {

    float* deviceOutputImageData;
    float* deviceConvOutputImageData;

    int gridSize = imageHeight * imageWidth / BLOCKSIZE;
    int convolutionChannels = 1;

    float numberBlockXTiling = (float)imageWidth / TILE_WIDTH;
    float numberBlockYTiling = (float)imageHeight / TILE_WIDTH;

    int numberBlockX = ceil(numberBlockXTiling);
    int numberBlockY = ceil(numberBlockYTiling);

    dim3 dimGrid(numberBlockX, numberBlockY);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);


    hipMalloc((void**)&deviceOutputImageData, imageWidth * imageHeight *
        sizeof(float));

    hipMalloc((void**)&deviceConvOutputImageData, imageWidth * imageHeight *
        sizeof(float));

    hipMemset(deviceOutputImageData, 0, imageWidth * imageHeight * sizeof(float));


    addMatrices << <gridSize, BLOCKSIZE >> > (deviceInputImageData, deviceOutputImageData,
        imageChannels, imageWidth, imageHeight);

    //print(deviceOutputImageData, imageHeight, imageWidth);

    clearMemory(deviceInputImageData, imageWidth * imageHeight *
        imageChannels * sizeof(float));


    prepareInputKernel << <dimGrid, dimBlock >> > (deviceOutputImageData, deviceMaskData, deviceConvOutputImageData,
        convolutionChannels, imageWidth, imageHeight);

    
    clearMemory(deviceOutputImageData, imageWidth * imageHeight * sizeof(float));

    return deviceConvOutputImageData;
}


int main() {

    int imageChannels = 3;
    int imageHeight = 8;
    int imageWidth = 8;

    float* hostInputImageData;
    float* deviceInputImageData;
    float* deviceOutputImageData;

    hostInputImageData = new float[imageHeight * imageWidth * imageChannels];

    // call only once in the main
    float hostMaskData[maskRows * maskCols];
    for (int i = 0; i < maskRows * maskCols; i++)
    {
        hostMaskData[i] = 1.0 / (maskRows * maskCols);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(deviceMaskData), hostMaskData, maskRows * maskCols * sizeof(float));

    for (int i = 0; i < imageChannels * imageHeight * imageWidth; i++) {
        hostInputImageData[i] = float((float)rand() / (RAND_MAX));
    }

    for (int i = 0; i < imageChannels * imageHeight * imageWidth; i++) {
        printf("%f, ", hostInputImageData[i]);
    }

    std::cout << "\n" << std::endl;

    hipMalloc((void**)&deviceInputImageData, imageWidth * imageHeight *
        imageChannels * sizeof(float));

    hipMemcpy(deviceInputImageData, hostInputImageData,
        imageWidth * imageHeight * imageChannels * sizeof(float),
        hipMemcpyHostToDevice);
    
    deviceOutputImageData = prepareInput(deviceInputImageData, imageHeight, imageWidth, imageChannels);
         
    hipMemset(deviceInputImageData, 0, imageWidth * imageHeight *
        imageChannels * sizeof(float));
    hipMemset(deviceOutputImageData, 0, imageWidth * imageHeight *
        sizeof(float));

    free(hostInputImageData);
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);

}